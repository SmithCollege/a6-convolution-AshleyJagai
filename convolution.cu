#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void naiveConvolution(float *input, float *output, float *mask, int inputSize, int maskSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int halfMaskSize = maskSize / 2;

    if (idx < inputSize) {
        float sum = 0.0f;
        for (int j = -halfMaskSize; j <= halfMaskSize; j++) {
            int inputIdx = idx + j;
            if (inputIdx >= 0 && inputIdx < inputSize) {
                sum += input[inputIdx] * mask[j + halfMaskSize];
            }
        }
        output[idx] = sum;
    }
}

void runNaiveConvolution(float *input, float *output, float *mask, int inputSize, int maskSize) {
    float *d_input, *d_output, *d_mask;
    hipMalloc(&d_input, inputSize * sizeof(float));
    hipMalloc(&d_output, inputSize * sizeof(float));
    hipMalloc(&d_mask, maskSize * sizeof(float));

    hipMemcpy(d_input, input, inputSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, maskSize * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (inputSize + blockSize - 1) / blockSize;
    naiveConvolution<<<numBlocks, blockSize>>>(d_input, d_output, d_mask, inputSize, maskSize);

    hipMemcpy(output, d_output, inputSize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);
}

__constant__ float d_mask[5]; // Assuming a mask size of 5 for this example

__global__ void constantMemoryConvolution(float *input, float *output, int inputSize, int maskSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int halfMaskSize = maskSize / 2;

    if (idx < inputSize) {
        float sum = 0.0f;
        for (int j = -halfMaskSize; j <= halfMaskSize; j++) {
            int inputIdx = idx + j;
            if (inputIdx >= 0 && inputIdx < inputSize) {
                sum += input[inputIdx] * d_mask[j + halfMaskSize];
            }
        }
        output[idx] = sum;
    }
}

void runConstantMemoryConvolution(float *input, float *output, float *mask, int inputSize, int maskSize) {
    float *d_input, *d_output;
    hipMalloc(&d_input, inputSize * sizeof(float));
    hipMalloc(&d_output, inputSize * sizeof(float));

    hipMemcpy(d_input, input, inputSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_mask), mask, maskSize * sizeof(float));

    int blockSize = 256;
    int numBlocks = (inputSize + blockSize - 1) / blockSize;
    constantMemoryConvolution<<<numBlocks, blockSize>>>(d_input, d_output, inputSize, maskSize);

    hipMemcpy(output, d_output, inputSize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
